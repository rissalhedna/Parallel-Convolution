#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <string.h>
#include <sys/time.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define DEFAULT_ITERATIONS 1
#define TILE_WIDTH 6

__global__ void convolve_cuda(int *sub_grid, int *new_grid, int nrows, int DIM, int *kernel, int kernel_dim)
{
    int num_pads = (kernel_dim - 1) / 2;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int row = blockIdx.x * blockDim.x + tx;
    int col = blockIdx.y * blockDim.y + ty;

    __shared__ int tile[TILE_WIDTH][TILE_WIDTH];

    if (row < nrows && col < nrows)
    {
        tile[ty][tx] = sub_grid[row * DIM + col];
    }
    else
    {
        tile[ty][tx] = 0;
    }
    __syncthreads();

    if (row < nrows && col < nrows && tx < TILE_WIDTH - num_pads && ty < TILE_WIDTH - num_pads)
    {
        int counter = 0;

        for (int j = 1; j < (num_pads + 1); j++)
        {
            counter += tile[ty + j][tx] * kernel[(((kernel_dim - 1) * (kernel_dim + 1)) / 2) + j * kernel_dim];
            counter += tile[ty - j][tx] * kernel[(((kernel_dim - 1) * (kernel_dim + 1)) / 2) - j * kernel_dim];
        }
        counter += tile[ty][tx] * kernel[(((kernel_dim - 1) * (kernel_dim + 1)) / 2)];

        new_grid[row * nrows + col] = counter;
    }
}


int *check_cuda(int *sub_grid, int nrows, int DIM, int *kernel, int kernel_dim, int num_iterations)
{
    int *new_grid = (int *)malloc(nrows * nrows * sizeof(int));
    int *d_sub_grid, *d_new_grid, *d_kernel;

    hipMalloc(&d_sub_grid, (DIM + (kernel_dim - 1)) * (DIM + (kernel_dim - 1)) * sizeof(int));
    hipMalloc(&d_kernel, kernel_dim * kernel_dim * sizeof(int));
    hipMalloc(&d_new_grid, nrows * nrows * sizeof(int));

    hipMemcpy(d_sub_grid, sub_grid, (DIM + (kernel_dim - 1)) * (DIM + (kernel_dim - 1)) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, kernel, kernel_dim * kernel_dim * sizeof(int), hipMemcpyHostToDevice);

    dim3 block_size(TILE_WIDTH, TILE_WIDTH);
    dim3 num_blocks((nrows + block_size.x - 1) / block_size.x, (nrows + block_size.y - 1) / block_size.y);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    for (int iter = 0; iter < num_iterations; iter++)
    {
        convolve_cuda<<<num_blocks, block_size>>>(d_sub_grid, d_new_grid, nrows, DIM + (kernel_dim - 1), d_kernel, kernel_dim);

        int *tmp = d_sub_grid;
        d_sub_grid = d_new_grid;
        d_new_grid = tmp;
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float elapsed_time;
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf("Execution time: %f s\n", elapsed_time / 100);

    hipMemcpy(new_grid, d_sub_grid +    ((kernel_dim - 1) / 2) * (DIM + (kernel_dim - 1)) + ((kernel_dim - 1) / 2), nrows * nrows * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_sub_grid);
    hipFree(d_kernel);

    return new_grid;
}

int main(int argc, char **argv)
{
    if (argc < 3)
    {
        printf("Usage: ./convolve <DIM> <NUM_ITERATIONS>\n");
        exit(1);
    }
    int DIM = atoi(argv[1]);
    int num_iterations = atoi(argv[2]);
    int kernel_dim = 5;
    int *sub_grid = (int *)malloc((DIM + (kernel_dim - 1)) * (DIM + (kernel_dim - 1)) * sizeof(int));
    int *kernel = (int *)malloc(kernel_dim * kernel_dim * sizeof(int));

    srand(42);

    for (int i = 0; i < DIM + (kernel_dim - 1); i++)
    {
        for (int j = 0; j < DIM + (kernel_dim - 1); j++)
        {
            if (i < (kernel_dim - 1) / 2 || i >= DIM + (kernel_dim - 1) - (kernel_dim - 1) / 2 || j < (kernel_dim - 1) / 2 || j >= DIM + (kernel_dim - 1) - (kernel_dim - 1) / 2)
            {
                sub_grid[i * (DIM + (kernel_dim - 1)) + j] = 0;
            }
            else
            {
                sub_grid[i * (DIM + (kernel_dim - 1)) + j] = rand() % 100;
            }
        }
    }


    int temp[] = {1, 4, 7, 4, 1, 4, 16, 26, 16, 4, 7, 26, 41, 26, 7, 4, 16, 26, 16, 4, 1, 4, 7, 4, 1};
    memcpy(kernel, temp, kernel_dim * kernel_dim * sizeof(int));

    int *result = check_cuda(sub_grid, DIM, DIM + (kernel_dim - 1), kernel, kernel_dim, num_iterations);
    // for (int i = 0; i < DIM; i++)
    // {
    //     for (int j = 0; j < DIM; j++)
    //     {
    //         printf("%d ", result[i * DIM + j]);
    //     }
    //     printf("\n");
    // }

    free(sub_grid);
    free(kernel);
    free(result);

    return 0;
}

